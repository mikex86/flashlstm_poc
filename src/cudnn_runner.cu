#include "cudnn_runner.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipDNN.h>
#include <cudnn_ops.h>
#include <cudnn_adv.h>

#include <cstdint>
#include <cstdio>
#include <vector>

#include "nvtx_profinst.h"

namespace {

using CudnnRnnForwardFn = hipdnnStatus_t (*)(hipdnnHandle_t,
                                            hipdnnRNNDescriptor_t,
                                            cudnnForwardMode_t,
                                            const int32_t*,
                                            cudnnRNNDataDescriptor_t,
                                            const void*,
                                            cudnnRNNDataDescriptor_t,
                                            void*,
                                            hipdnnTensorDescriptor_t,
                                            const void*,
                                            void*,
                                            hipdnnTensorDescriptor_t,
                                            const void*,
                                            void*,
                                            size_t,
                                            const void*,
                                            size_t,
                                            void*,
                                            size_t,
                                            void*);

constexpr CudnnRnnForwardFn kCudnnRnnForward = &cudnnRNNForward;

}  // namespace

int initialize_cudnn() {
    NVTX_SCOPED_RANGE("cuDNN::initialize");
    hipdnnHandle_t handle = nullptr;
    hipdnnStatus_t status = hipdnnCreate(&handle);
    if (status != HIPDNN_STATUS_SUCCESS) {
        std::fprintf(stderr,
                     "cuDNN error %s (%d) in hipdnnCreate during initialize_cudnn\n",
                     hipdnnGetErrorString(status),
                     static_cast<int>(status));
        return static_cast<int>(status);
    }
    status = hipdnnDestroy(handle);
    if (status != HIPDNN_STATUS_SUCCESS) {
        std::fprintf(stderr,
                     "cuDNN error %s (%d) in hipdnnDestroy during initialize_cudnn\n",
                     hipdnnGetErrorString(status),
                     static_cast<int>(status));
        return static_cast<int>(status);
    }
    return 0;
}

int run_cudnn_lstm(const float* x_host,
                   const float* h0_host,
                   const float* c0_host,
                   float* y_host,
                   float* hy_host,
                   float* cy_host,
                   const std::size_t seq_len,
                   const std::size_t batch,
                   const std::size_t input_size,
                   const std::size_t hidden_size) {
    int status = 0;

    NVTX_SCOPED_RANGE("run_cudnn_lstm");

    const std::size_t x_bytes = seq_len * batch * input_size * sizeof(float);
    const std::size_t y_bytes = seq_len * batch * hidden_size * sizeof(float);
    const std::size_t state_bytes = batch * hidden_size * sizeof(float);
    const std::uint64_t seed = 1234ULL;

    const int tensor_dims = 3;
    int state_dims[tensor_dims] = {1,
                                   static_cast<int>(batch),
                                   static_cast<int>(hidden_size)};
    int state_strides[tensor_dims] = {static_cast<int>(batch * hidden_size),
                                      static_cast<int>(hidden_size),
                                      1};

    std::vector<int> seq_lengths_host(batch, static_cast<int>(seq_len));
    std::vector<int32_t> seq_lengths_device_host(batch, static_cast<int32_t>(seq_len));

    size_t dropout_states_bytes = 0;
    size_t workspace_bytes = 0;
    size_t reserve_space_bytes = 0;
    size_t weight_space_bytes = 0;

    hipdnnHandle_t handle = nullptr;
    hipdnnDropoutDescriptor_t dropout_desc = nullptr;
    hipdnnRNNDescriptor_t rnn_desc = nullptr;
    cudnnRNNDataDescriptor_t x_data_desc = nullptr;
    cudnnRNNDataDescriptor_t y_data_desc = nullptr;
    hipdnnTensorDescriptor_t h_desc = nullptr;
    hipdnnTensorDescriptor_t c_desc = nullptr;

    float* d_x = nullptr;
    float* d_y = nullptr;
    float* d_hx = nullptr;
    float* d_cx = nullptr;
    float* d_hy = nullptr;
    float* d_cy = nullptr;
    void* d_weight_space = nullptr;
    void* d_workspace = nullptr;
    void* d_reserve_space = nullptr;
    void* d_dropout_states = nullptr;
    int32_t* d_seq_lengths = nullptr;

    auto check_cuda = [&](hipError_t err, const char* what) {
        if (err != hipSuccess) {
            std::fprintf(stderr,
                         "CUDA error %s (%d) in %s\n",
                         hipGetErrorString(err),
                         static_cast<int>(err),
                         what);
            status = static_cast<int>(err);
            return false;
        }
        return true;
    };

    auto check_cudnn = [&](hipdnnStatus_t err, const char* what) {
        if (err != HIPDNN_STATUS_SUCCESS) {
            std::fprintf(stderr,
                         "cuDNN error %s (%d) in %s\n",
                         hipdnnGetErrorString(err),
                         static_cast<int>(err),
                         what);
            status = static_cast<int>(err);
            return false;
        }
        return true;
    };

    auto check_cuda_wrap = [&](hipError_t err, const char* what) {
        return check_cuda(err, what);
    };
    auto check_cudnn_wrap = [&](hipdnnStatus_t err, const char* what) {
        return check_cudnn(err, what);
    };

#define CUDA_CALL(expr)                      \
    do {                                     \
        if (!check_cuda_wrap((expr), #expr)) \
            goto cleanup;                    \
    } while (0)

#define CUDNN_CALL(expr)                     \
    do {                                     \
        if (!check_cudnn_wrap((expr), #expr))\
            goto cleanup;                    \
    } while (0)

    CUDNN_CALL(hipdnnCreate(&handle));
    CUDNN_CALL(hipdnnCreateDropoutDescriptor(&dropout_desc));
    CUDNN_CALL(hipdnnCreateRNNDescriptor(&rnn_desc));
    CUDNN_CALL(cudnnCreateRNNDataDescriptor(&x_data_desc));
    CUDNN_CALL(cudnnCreateRNNDataDescriptor(&y_data_desc));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&h_desc));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&c_desc));

    CUDA_CALL(hipMalloc(&d_x, x_bytes));
    CUDA_CALL(hipMalloc(&d_y, y_bytes));
    CUDA_CALL(hipMalloc(&d_hx, state_bytes));
    CUDA_CALL(hipMalloc(&d_cx, state_bytes));
    CUDA_CALL(hipMalloc(&d_hy, state_bytes));
    CUDA_CALL(hipMalloc(&d_cy, state_bytes));
    CUDA_CALL(hipMalloc(&d_seq_lengths, seq_lengths_device_host.size() * sizeof(int32_t)));

    CUDA_CALL(hipMemcpy(d_x, x_host, x_bytes, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_hx, h0_host, state_bytes, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_cx, c0_host, state_bytes, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_seq_lengths,
                         seq_lengths_device_host.data(),
                         seq_lengths_device_host.size() * sizeof(int32_t),
                         hipMemcpyHostToDevice));
    CUDA_CALL(hipMemset(d_y, 0, y_bytes));
    CUDA_CALL(hipMemset(d_hy, 0, state_bytes));
    CUDA_CALL(hipMemset(d_cy, 0, state_bytes));

    CUDNN_CALL(hipdnnDropoutGetStatesSize(handle, &dropout_states_bytes));
    if (dropout_states_bytes > 0) {
        CUDA_CALL(hipMalloc(&d_dropout_states, dropout_states_bytes));
        CUDA_CALL(hipMemset(d_dropout_states, 0, dropout_states_bytes));
    }

    CUDNN_CALL(hipdnnSetDropoutDescriptor(dropout_desc,
                                         handle,
                                         0.0f,
                                         d_dropout_states,
                                         dropout_states_bytes,
                                         seed));

    CUDNN_CALL(cudnnSetRNNDescriptor_v8(rnn_desc,
                                        HIPDNN_RNN_ALGO_STANDARD,
                                        HIPDNN_LSTM,
                                        HIPDNN_RNN_WITH_BIAS,
                                        HIPDNN_UNIDIRECTIONAL,
                                        HIPDNN_LINEAR_INPUT,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_DATA_FLOAT,
                                        CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION,
                                        static_cast<int32_t>(input_size),
                                        static_cast<int32_t>(hidden_size),
                                        static_cast<int32_t>(hidden_size),
                                        1,
                                        dropout_desc,
                                        0));

    CUDNN_CALL(cudnnBuildRNNDynamic(handle,
                                    rnn_desc,
                                    static_cast<int>(batch)));

    CUDNN_CALL(cudnnSetRNNDataDescriptor(x_data_desc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_UNPACKED,
                                         static_cast<int>(seq_len),
                                         static_cast<int>(batch),
                                         static_cast<int>(input_size),
                                         seq_lengths_host.data(),
                                         nullptr));

    CUDNN_CALL(cudnnSetRNNDataDescriptor(y_data_desc,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_UNPACKED,
                                         static_cast<int>(seq_len),
                                         static_cast<int>(batch),
                                         static_cast<int>(hidden_size),
                                         seq_lengths_host.data(),
                                         nullptr));

    CUDNN_CALL(hipdnnSetTensorNdDescriptor(h_desc,
                                          HIPDNN_DATA_FLOAT,
                                          tensor_dims,
                                          state_dims,
                                          state_strides));
    CUDNN_CALL(hipdnnSetTensorNdDescriptor(c_desc,
                                          HIPDNN_DATA_FLOAT,
                                          tensor_dims,
                                          state_dims,
                                          state_strides));

    CUDNN_CALL(cudnnGetRNNTempSpaceSizes(handle,
                                         rnn_desc,
                                         CUDNN_FWD_MODE_INFERENCE,
                                         x_data_desc,
                                         &workspace_bytes,
                                         &reserve_space_bytes));
    CUDNN_CALL(cudnnGetRNNWeightSpaceSize(handle,
                                          rnn_desc,
                                          &weight_space_bytes));

    if (workspace_bytes > 0) {
        CUDA_CALL(hipMalloc(&d_workspace, workspace_bytes));
        CUDA_CALL(hipMemset(d_workspace, 0, workspace_bytes));
    }
    if (reserve_space_bytes > 0) {
        CUDA_CALL(hipMalloc(&d_reserve_space, reserve_space_bytes));
        CUDA_CALL(hipMemset(d_reserve_space, 0, reserve_space_bytes));
    }
    if (weight_space_bytes > 0) {
        CUDA_CALL(hipMalloc(&d_weight_space, weight_space_bytes));
        CUDA_CALL(hipMemset(d_weight_space, 0, weight_space_bytes));
    }

    {
        NVTX_SCOPED_RANGE("cuDNN::cudnnRNNForward");
        CUDNN_CALL(kCudnnRnnForward(handle,
                                    rnn_desc,
                                    CUDNN_FWD_MODE_INFERENCE,
                                    d_seq_lengths,
                                    x_data_desc,
                                    d_x,
                                    y_data_desc,
                                    d_y,
                                    h_desc,
                                    d_hx,
                                    d_hy,
                                    c_desc,
                                    d_cx,
                                    d_cy,
                                    weight_space_bytes,
                                    d_weight_space,
                                    workspace_bytes,
                                    d_workspace,
                                    reserve_space_bytes,
                                    d_reserve_space));
    }

    CUDA_CALL(hipMemcpy(y_host, d_y, y_bytes, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(hy_host, d_hy, state_bytes, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(cy_host, d_cy, state_bytes, hipMemcpyDeviceToHost));

#undef CUDA_CALL
#undef CUDNN_CALL

cleanup:
    if (d_reserve_space != nullptr) {
        hipFree(d_reserve_space);
    }
    if (d_workspace != nullptr) {
        hipFree(d_workspace);
    }
    if (d_weight_space != nullptr) {
        hipFree(d_weight_space);
    }
    if (d_dropout_states != nullptr) {
        hipFree(d_dropout_states);
    }
    if (d_seq_lengths != nullptr) {
        hipFree(d_seq_lengths);
    }
    if (d_cy != nullptr) {
        hipFree(d_cy);
    }
    if (d_hy != nullptr) {
        hipFree(d_hy);
    }
    if (d_cx != nullptr) {
        hipFree(d_cx);
    }
    if (d_hx != nullptr) {
        hipFree(d_hx);
    }
    if (d_y != nullptr) {
        hipFree(d_y);
    }
    if (d_x != nullptr) {
        hipFree(d_x);
    }

    if (c_desc != nullptr) {
        hipdnnDestroyTensorDescriptor(c_desc);
    }
    if (h_desc != nullptr) {
        hipdnnDestroyTensorDescriptor(h_desc);
    }
    if (y_data_desc != nullptr) {
        cudnnDestroyRNNDataDescriptor(y_data_desc);
    }
    if (x_data_desc != nullptr) {
        cudnnDestroyRNNDataDescriptor(x_data_desc);
    }
    if (rnn_desc != nullptr) {
        hipdnnDestroyRNNDescriptor(rnn_desc);
    }
    if (dropout_desc != nullptr) {
        hipdnnDestroyDropoutDescriptor(dropout_desc);
    }
    if (handle != nullptr) {
        hipdnnDestroy(handle);
    }

    return status;
}
